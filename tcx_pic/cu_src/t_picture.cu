#include "hip/hip_runtime.h"
#include "../include/t_picture.h"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <iostream>

#define __cu_check_ret(_err, ...)\
    if(_err != hipSuccess){\
        std::cerr << "CUDA Runtime Error at: " << __FILE__ << ":" << __LINE__\
                    << hipGetErrorString(err) << " " << __FUNCTION__ << std::endl;\
        return __VA_ARGS__; \
    }(void)0

#define __cu_check(_err,...)\
    if(_err != hipSuccess){\
        std::cerr << "CUDA Runtime Error at: " << __FILE__ << ":" << __LINE__\
                    << hipGetErrorString(err) << " " << __FUNCTION__ << std::endl;\
        __VA_ARGS__; \
    }(void)0

__global__ static void g_t_pic_rgb_to_gray(char* buf, int line_byte_size){
    char* cur_buf = buf + threadIdx.x * line_byte_size;
    for(int i = 0 ; i < line_byte_size;i+=3){
        int gray = (cur_buf[i]*19595 + cur_buf[i+1]*38469 + cur_buf[i+2]*7472) >> 16;
        cur_buf[i] = gray;
        cur_buf[i+1] = gray;
        cur_buf[i+2] = gray;
    }
}

__global__ static void g_t_pic_bgr_to_gray(char* buf, int line_byte_size) {
    char* cur_buf = buf + threadIdx.x * line_byte_size;
    for (int i = 0; i < line_byte_size; i += 3) {
        int gray = (cur_buf[i+2] * 19595 + cur_buf[i + 1] * 38469 + cur_buf[i] * 7472) >> 16;
        cur_buf[i] = gray;
        cur_buf[i + 1] = gray;
        cur_buf[i + 2] = gray;
    }
}

__global__ static void g_t_pic_rgba_to_gray(char* buf, int line_byte_size) {
    char* cur_buf = buf + threadIdx.x * line_byte_size;
    for (int i = 0; i < line_byte_size; i += 4) {
        int gray = (cur_buf[i] * 19595 + cur_buf[i + 1] * 38469 + cur_buf[i + 2] * 7472) >> 16;
        cur_buf[i] = gray;
        cur_buf[i + 1] = gray;
        cur_buf[i + 2] = gray;
    }
}


int t_pic_where(){
    return -1;
}

int t_pic_load_pics(t_pic* pic, const char* path){
    if (!pic) return T_PIC_ERR;
    if (!path) return T_PIC_ERR;

    hipError_t err;
    BITMAP bm;
    pic->img = new CImage();
    CImage& img = *pic->img;
    HRESULT hr = img.Load(path);


    
    if (!SUCCEEDED(hr)) return T_PIC_ERR;
    HBITMAP hbmp = img;
    if (!GetObject(hbmp, sizeof(bm), &bm)) {
        img.Destroy();
        return T_PIC_ERR;
    }

    void* bt1 = bm.bmBits;
    void* bt2 = img.GetBits();
    
    
    int each_pixel_size = bm.bmWidthBytes / bm.bmWidth;
    int total_size = bm.bmWidthBytes * bm.bmHeight;

    err = hipMalloc( & pic->data, total_size);
    __cu_check_ret(err,T_PIC_ERR);

    err = hipMemcpy(pic->data, bm.bmBits, total_size, hipMemcpyHostToDevice);
    __cu_check_ret(err,T_PIC_ERR);


    pic->height = bm.bmHeight;
    pic->width = bm.bmWidth;
    pic->color_space = t_pic_bgr;

    return T_PIC_OK;
}

void t_pic_release(t_pic* pic){
    if (!pic)return;
    if(pic->data){
        hipError_t err = hipFree(pic->data);
        __cu_check(err);// free error
        pic->data = nullptr;
    }
    if (pic->img) {
        pic->img->Destroy();
        delete pic->img;
        pic->img = nullptr;
    }
}

int t_pic_to_gray(t_pic* pic){
    
    switch(pic->color_space){
        int line_byte_size;
        case t_pic_rgb:{
            line_byte_size = 3 * pic->width;
            g_t_pic_rgb_to_gray<<<1,pic->height>>>(pic->data, line_byte_size);
            break;
        }
        case t_pic_bgr: {
            line_byte_size = 3 * pic->width;
            g_t_pic_bgr_to_gray <<<1, pic->height >>> (pic->data, line_byte_size);
            break;
        }
        case t_pic_hsl: {
            line_byte_size = 3 * pic->width;
            break;
        }
        case t_pic_hsv: {
            line_byte_size = 3 * pic->width;
            break;
        }
        case t_pic_rgba: {
            line_byte_size = 4* pic->width;
            g_t_pic_rgba_to_gray << <1, pic->height >> > (pic->data, line_byte_size);
            break;
        }
    }
    

    hipDeviceSynchronize();
    auto err = hipGetLastError();
    __cu_check_ret(err, T_PIC_ERR);

    return T_PIC_OK;
}


int t_pic_save(t_pic* pic, const char* path) {
    if (!pic) return T_PIC_ERR;
    if (!path) return T_PIC_ERR;

    CImage& bm = *pic->img;
    int each_size = pic->color_space == t_pic_rgba? 4:3;
    int total_size = pic->width* pic->height* each_size;
    char* dist = (char*)malloc(total_size);
    hipError_t err = hipMemcpy(dist, pic->data,total_size, hipMemcpyDeviceToHost);
    
    __cu_check(err, T_PIC_ERR);

    free(dist);

    HRESULT hr = bm.Save(path);
    if (!SUCCEEDED(hr)) return T_PIC_ERR;


    return T_PIC_OK;
}
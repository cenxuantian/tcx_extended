#include "hip/hip_runtime.h"
#include "../include/t_picture.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>


#include <iostream>

#define __cu_check_ret(_err, ...)\
    if(_err != hipSuccess){\
        std::cerr << "CUDA Runtime Error at: " << __FILE__ << ":" << __LINE__\
                    << hipGetErrorString(err) << " " << __FUNCTION__ << std::endl;\
        return __VA_ARGS__; \
    }(void)0

#define __cu_check(_err,...)\
    if(_err != hipSuccess){\
        std::cerr << "CUDA Runtime Error at: " << __FILE__ << ":" << __LINE__\
                    << hipGetErrorString(err) << " " << __FUNCTION__ << std::endl;\
        __VA_ARGS__; \
    }(void)0

__global__ static void g_t_pic_rgb_to_gray(t_pics* pics){
    int size = pics->height * pics->width * 3;
    char* cur_buf = pics->data + threadIdx.x * size;
    for(int i = 0 ; i < size;i+=3){
        int gray = (cur_buf[i]*19595 + cur_buf[i+1]*38469 + cur_buf[i+2]*7472) >> 16;
        cur_buf[i] = gray;
        cur_buf[i+1] = gray;
        cur_buf[i+2] = gray;
    }
}

__global__ static void g_t_pic_rgba_to_gray(t_pics* pics) {
    int size = pics->height * pics->width * 4;
    char* cur_buf = pics->data + threadIdx.x * size;
    for (int i = 0; i < size; i += 4) {
        int gray = (cur_buf[i] * 19595 + cur_buf[i + 1] * 38469 + cur_buf[i + 2] * 7472) >> 16;
        cur_buf[i] = gray;
        cur_buf[i + 1] = gray;
        cur_buf[i + 2] = gray;
    }
}


int t_pic_where(){
    return -1;
}

int t_pic_load_pics(t_pics** pics,const char* path){
    hipError_t err = hipMalloc(pics,sizeof(t_pics));
    __cu_check_ret(err,T_PIC_ERR);

    (*pics)->height = 0;
    (*pics)->width = 0;
    (*pics)->count = 1;
    (*pics)->data = nullptr;

    return T_PIC_OK;
}

void t_pic_release(t_pics* pics){
    if (!pics)return;
    if(pics->data){
        hipError_t err = hipFree(pics->data);
        __cu_check(err);// free error
    }
    hipError_t err = hipFree(pics);
    __cu_check_ret(err);
}

int t_pic_to_gray(t_pics* pics){
    switch(pics->color_space){
        case t_pic_rgb:{
            g_t_pic_rgb_to_gray<<<1,pics->count>>>(pics);
            break;
        }
        case t_pic_hsl: {
            break;
        }
        case t_pic_hsv: {
            break;
        }
        case t_pic_rgba: {
            g_t_pic_rgba_to_gray <<<1, pics->count >>> (pics);
            break;
        }
    }
    

    hipDeviceSynchronize();
    auto err = hipGetLastError();
    __cu_check_ret(err, T_PIC_ERR);

    return T_PIC_OK;
}

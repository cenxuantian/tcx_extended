#include "hip/hip_runtime.h"
#include "../include/t_picture.h"
#include <hip/hip_runtime.h>


__global__ void g_t_pic_rgb_to_gray(t_pics* pics){
    int size = pics->height * pics->width * 3;
    char* cur_buf = pics->data + threadIdx.x * size;
    for(int i = 0 ; i < size;i+=3){
        int gray = (cur_buf[i]*19595 + cur_buf[i+1]*38469 + cur_buf[i+2]*7472) >> 16;
        cur_buf[i] = gray;
        cur_buf[i+1] = gray;
        cur_buf[i+2] = gray;
    }
}


int t_pic_where(){
    return -1;
}

int t_pic_load_pics(t_pics** pics,const char* path){
    hipMalloc(pics,sizeof(t_pics));
    (*pics)->height = 0;
    (*pics)->width = 0;
    (*pics)->count = 1;
    (*pics)->data = NULL;
    return T_PIC_OK;
}

void t_pic_release(t_pics* pics){
    hipFree(pics);
}

int t_pic_to_gray(t_pics* pics){
    switch(pics->color_space){
        case t_pic_rgb:{
            g_t_pic_rgb_to_gray<<<1,pics->count>>>(pics);
            break;
        }
    }
    return T_PIC_OK;
}

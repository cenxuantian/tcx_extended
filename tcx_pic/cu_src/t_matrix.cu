#include "hip/hip_runtime.h"
#include "../include/t_matrix.h"

#include "t_cuda_def.cuh"



int t_mat_create(t_mat * mat) {
    int total_size = mat->width * mat->height * mat->each_size;
    auto err = hipMalloc(&mat->data, total_size);
    __cu_check_ret(err, T_MAT_ERR);
    err = hipMemset(mat->data, 0, total_size);
    __cu_check_ret(err, T_MAT_ERR);
    return T_MAT_OK;
}
void t_mat_release(t_mat* mat) {
    if (mat->data) {
        hipFree(mat->data);
        mat->data = NULL;
    }
    mat->height = 0;
    mat->width = 0;
}

// add
__global__ void g_matmat_add_int(void* __restrict__ _1, void* __restrict__ _2, int width) {
    int* _a = (int*)_1;
    int* _b = (int*)_2;

    int cur_row = blockIdx.x;
    int cur_cow = threadIdx.x;

    loop:
    _a[cur_row * width + cur_cow] += _b[cur_row * width + cur_cow];

    cur_cow += blockDim.x;
    if (cur_cow<width) {
        goto loop;
    }
    return;

}
__global__ void g_matmat_add_double(void* __restrict__ _1, void* __restrict__ _2, int width) {
    double* _a = (double*)_1;
    double* _b = (double*)_2;

    int cur_row = blockIdx.x;
    int cur_cow = threadIdx.x;

loop:
    _a[cur_row * width + cur_cow] += _b[cur_row * width + cur_cow];

    cur_cow += blockDim.x;
    if (cur_cow < width) {
        goto loop;
    }
    return;

}
__global__ void g_matnum_multi_int(void* _1, double num, int width) {
    int* _a = (int*)_1;

    int cur_row = blockIdx.x;
    int cur_cow = threadIdx.x;

loop:
    _a[cur_row * width + cur_cow] *= num;

    cur_cow += blockDim.x;
    if (cur_cow < width) {
        goto loop;
    }
    return;
}
__global__ void g_matnum_multi_double(void* _1, double num, int width) {

    double* _a = (double*)_1;

    int cur_row = blockIdx.x;
    int cur_cow = threadIdx.x;

loop:
    _a[cur_row * width + cur_cow] *= num;

    cur_cow += blockDim.x;
    if (cur_cow < width) {
        goto loop;
    }
    return;
}
__global__ void g_mat_add_num_int(void* _1, double num, int max_idx) {
    int i = t_cuIdx;
    int max_thread = t_cuMaxThreadCount;
    int* data = (int*)_1;
    
    while (i < max_idx) {
        data[i] += num;
        i += max_thread;
    }
}
__global__ void g_mat_add_num_double(void* _1, double num, int max_idx) {
    int i = t_cuIdx;
    int max_thread = t_cuMaxThreadCount;
    double* data = (double*)_1;

    while (i < max_idx) {
        data[i] += num;
        i += max_thread;
    }
}
int t_mat_add_mat(t_mat* _1, t_mat* _2) {
    if (!(_1->width == _2->width && _1->height == _2->height && _1->each_size == _2->each_size)) {
        return T_MAT_ERR;
    }
    if (_1->each_size == sizeof(int)) {
        g_matmat_add_int << < _1->height, min(_1->width, 1024) >> > (_1->data, _2->data, _1->width);
    }
    else if (_1->each_size == sizeof(double)) {
        g_matmat_add_double << < _1->height, min(_1->width, 1024) >> > (_1->data, _2->data, _1->width);
    }
    return T_MAT_OK;
}
int t_mat_add_num(t_mat* _1, double num) {
    int max_idx = _1->height * _1->width;
    int block, thread;
    t_cu_get_launch_arg(max_idx, &block, &thread);
    if (_1->each_size == sizeof(int)) {
        g_mat_add_num_int << <block, thread >>> (_1->data, num, max_idx);
    }
    else if (_1->each_size == sizeof(double)) {
        g_mat_add_num_double << <block, thread >>> (_1->data, num, max_idx);
    }
    
    return T_MAT_OK;
}
int t_mat_multiply_num(t_mat* _1, double num) {
    if (_1->each_size == sizeof(int)) {
        g_matnum_multi_int << <_1->height, min(_1->width, 1024) >> > (_1->data, num, _1->width);
    }
    else if (_1->each_size == sizeof(double)) {
        g_matnum_multi_double << <_1->height, min(_1->width, 1024) >> > (_1->data, num, _1->width);
    }
    return T_MAT_OK;
}


int t_mat_transpos(t_mat* mat) {
    return T_MAT_OK;
}
int t_mat_clone(t_mat* dist, t_mat* src) {
    dist->each_size = src->each_size;
    dist->height = src->each_size;
    dist->width = src->width;
    int ret = t_mat_create(dist);
    if (ret == T_MAT_ERR) return ret;
    auto err = hipMemcpy(dist->data, src->data, dist->height * dist->width * dist->each_size, hipMemcpyDeviceToDevice);
    __cu_check(err, T_MAT_ERR);
    return T_MAT_OK;
}
int t_mat_inverse(t_mat* mat) {
    return T_MAT_OK;
}

int t_mat_get_int(t_mat* mat,int x, int y) {
    int ret;
    t_mat_get(mat, x, y, &ret, sizeof(ret));
    return ret;
}
double t_mat_get_double(t_mat* mat,int x, int y) {
    double ret;
    t_mat_get(mat, x, y, &ret, sizeof(ret));
    return ret;
}
int t_mat_get(t_mat* mat,int x, int y, void* out, int size_of_out) {
    auto err = hipMemcpy(out, (char*)(mat->data) + (y * mat->width + x * size_of_out), size_of_out, hipMemcpyDeviceToHost);
    __cu_check(err, T_MAT_ERR);
    return T_MAT_OK;
}
int t_mat_set_int(t_mat* mat,int x, int y, int val) {
    return t_mat_set(mat, x, y, &val, sizeof(val));
}
int t_mat_set_double(t_mat* mat,int x, int y, double val) {
    return t_mat_set(mat, x, y, &val, sizeof(val));
}
int t_mat_set(t_mat* mat,int x, int y, void* in, int size_of_in) {
    auto err = hipMemcpy((char*)(mat->data) + ((y * mat->width + x) * size_of_in), in, size_of_in, hipMemcpyHostToDevice);
    __cu_check(err, T_MAT_ERR);
    return T_MAT_OK;
}

__global__ void t_mat_print_int(int* data, int width,int height) {
    for (int i = 0; i < height;i++) {
        for (int j = 0; j < width; j++) {
            printf("%d ", data[i * width + j]);
        }
        printf("\n");
    }
}
__global__ void t_mat_print_double(double* data, int width, int height) {
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            printf("%d ", data[i * width + j]);
        }
        printf("\n");
    }
}
void t_mat_print(t_mat* mat) {
    if (mat->each_size == sizeof(int)) {
        t_mat_print_int<<<1,1>>>((int*)mat->data, mat->width, mat->height);
    }
    else if (mat->each_size == sizeof(double)) {
        t_mat_print_double<<<1,1>>>((double*)mat->data, mat->width, mat->height);
    }
}
void t_mat_synchronize() {
    hipDeviceSynchronize();
    auto err = hipGetLastError();
    __cu_check(err);
}









